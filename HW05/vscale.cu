#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vscale.cuh>

__global__ void vscale(const float *a, float *b, unsigned int n){
    int index = threadIdx.x+blockIdx.x*blockDim.x; 
    b[index] = a[index]*b[index];
}