#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include "reduce.cuh"

int main(int argc, char *argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " N threads_per_block\n";
        return 1;
    }

    unsigned int N = std::atoi(argv[1]);
    unsigned int threads_per_block = std::atoi(argv[2]);

    // Allocate and fill host array
    float *h_input = new float[N];
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    for (unsigned int i = 0; i < N; ++i) {
        h_input[i] = dist(gen);
    }

    // Allocate device memory
    float *d_input, *d_output;
    hipMalloc((void **)&d_input, N * sizeof(float));
    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

    unsigned int blocks = (N + threads_per_block * 2 - 1) / (threads_per_block * 2);
    hipMalloc((void **)&d_output, blocks * sizeof(float));

    // Start timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Call reduce function
    reduce(&d_input, &d_output, N, threads_per_block);

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // Copy result back to host
    float result;
    hipMemcpy(&result, d_input, sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    std::cout << "Sum: " << result << std::endl;
    std::cout << "Time: " << ms << " ms" << std::endl;

    // Cleanup
    delete[] h_input;
    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}